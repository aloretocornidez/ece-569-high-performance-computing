#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                \
  do                                                                 \
  {                                                                  \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
      return -1;                                                     \
    }                                                                \
  } while (0)

//@@ INSERT DEVICE CODE HERE
__global__ void grayScaleConversion(float *output, float *input, int height, int width)
{
  // Get the row and the column location in the image of the thread.
  int column = blockDim.x * blockIdx.x + threadIdx.x;
  int row = blockDim.y * blockIdx.y + threadIdx.y;

  // Boundary check condition.
  if (row < height && column < width)
  {
    // Set the index to access in the arrays
    int idx = row * width + column;

    // Get rgb values from the input image data array.
    float r = input[3 * idx];
    float g = input[3 * idx + 1];
    float b = input[3 * idx + 2];

    // Set the output array to a gray-scale value from RGB values.
    output[idx] = (float)(0.21 * r + 0.71 * g + 0.07 * b);
  }
}

// Also modify the main function to launch thekernel.
int main(int argc, char *argv[])
{
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  // Create 32*32 threads = 1024 threads
  dim3 threadsPerBlock(32, 32);

  // Divides the image size into the number of blocks that need to be launched.
  dim3 blocksPerGrid(ceil(imageWidth / 32.0), ceil(imageHeight / 32.0));

  // Call the grayscale conversion kernel.
  grayScaleConversion<<<blocksPerGrid, threadsPerBlock>>>(deviceOutputImageData, deviceInputImageData, imageHeight, imageWidth);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
