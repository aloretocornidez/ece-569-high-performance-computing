#include "hip/hip_runtime.h"
/*
load the cuda module:
$ module load cuda11/11.0

to compile: 
$ nvcc -o myadd madd.cu

to execute: 
$ ./myadd array_dimension

Receives array dimension as command line parameter (2048 suggested size)

You will implement three versions of matrix addition. 



Version-1) kernel_1t1e: each thread produces one output matrix
Version-2) kernel_1t1r: each thread produces 1 output row
Version-3) kernel_1t1c: each thread produces 1 output column

Assume that inputs are square matrices. 

In this template main() handles the following: 
-Initialize input matrices A and B, 
-Allocate memory on device for input (A and B) and output (C)
-transfer A and B arrays to device
-Set up grid of blocks where block size is fixed to 32x32
-Launch each kernel twice, collect timing information, take average
-transfer data from device to host
-print first 5 output elements for functional verification across 3 kernels.

You should not make any change in the main(). 

Only fill in the functionality for the three kernels declared below
starting at line 95. 

Compare the execution times for the three kernels. 
Version-1 offers the highest degree of parallelism. It is expected to have 
shortest execution time. How about versions 2 and 3? Which one is faster? Why?

Check your implementation against the solution provided in the same folder. 
Note that for version-2, there is an additional activity suggested for you to 
experiment with. 


*/


#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
//#include <time.h>

// CPU 
void randomArray(float *cpu_arrayA,float *cpu_arrayB, unsigned long SQWIDTH) {
	srand((unsigned) time(NULL));
	for(unsigned long i = 0; i<SQWIDTH*SQWIDTH; ++i){
		cpu_arrayA[i] = ((float)rand()/(float)(RAND_MAX)) * 100;
		cpu_arrayB[i] = ((float)rand()/(float)(RAND_MAX)) * 100;
		//printf("Matrx [%d][%d]: %.2f\n", i,j, cpu_array[i][j]);
	}
}

void printResults(float *h_matA, float *h_matB, float *h_matC,int SQWIDTH){
	printf("Matrix A:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matA[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
	printf("Matrix B:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matB[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
	printf("Matrix C:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matC[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
}

// Templates for the three kernels are proviced below. 
__global__ void kernel_1t1e(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: each thread produces one output matrix






}

__global__ void kernel_1t1r(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: each thread produces 1 output row







}

__global__ void kernel_1t1c(float *A, float *B, float *C, unsigned long WIDTH) {
     // To DO: each thread produces 1 output column








}



int main(int argc, char* argv[]) {
	// Memory specification
	
	if(argc<=1) {
        printf("You did not feed me array size\n");
        exit(1);
     }  //otherwise continue on our merry way....
    
	
	
	
	unsigned long SQWIDTH;
	
	 SQWIDTH = atoi(argv[1]);


	const size_t d_size = sizeof(float) * size_t(SQWIDTH*SQWIDTH);

	// Multiprocessing constants
	const dim3 threadsPerBlock(32,32); 	// Must not exceed 1024 (max thread per block)
	const dim3 blocksPerGrid(ceil(SQWIDTH/32.0),ceil(SQWIDTH/32.0));		// Number of blocks that will be used

	// CUDA TIME
	float ms;
	float avems = 0.0;
	hipEvent_t start,end;



	// Initialize host matrices
	//clock_t h_alloctime = clock();
	float *h_matA = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	float *h_matB = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	float *h_matC = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	randomArray(h_matA, h_matB, SQWIDTH);
	//printf("[**] CPU Allocation time for %dx%d matrix: %.2fsec \n",SQWIDTH,SQWIDTH,(double)(clock()-h_alloctime)/CLOCKS_PER_SEC );
	// Initialize device matrices
	float *d_matA, *d_matB, *d_matC;
	
	clock_t d_alloctime = clock();
	hipMalloc((void **) &d_matA, d_size);
	hipMalloc((void **) &d_matB, d_size);
	hipMalloc((void **) &d_matC, d_size);
	hipMemcpy(d_matA, h_matA, d_size, hipMemcpyHostToDevice);
	hipMemcpy(d_matB, h_matB, d_size, hipMemcpyHostToDevice); 
	//printf("[**] GPU Allocation time for %lux%lu matrix: %.2fsec \n",SQWIDTH,SQWIDTH,(double)(clock()-d_alloctime)/CLOCKS_PER_SEC );


	// Number of threads = SQWIDTH*SQWIDTH
	printf("[**] Starting kernel program 'kernel_1t1e' execution\n");
	for(int i = 0; i<2; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1e<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.2f\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 
        
        
          
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("Printing first five output elements only\n");
	for(int i=0; i< 5; i++){
		printf("%.2f	", h_matC[i]);}
		printf("\n");
	printf("[**] Average kernel execution time: %.2f.\n\n", avems/2.0);
	

	printf("[] Starting kernel program 'kernel_1t1r'.execution\n");
	avems = 0.0;
	for(int i = 0; i<2; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1r<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.2f\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("Printing first five output elements only\n");
		for(int i=0; i< 5; i++){
		printf("%.2f	", h_matC[i]);}
		printf("\n");
	printf("[**] Average kernel execution time: %.2f\n\n", avems/2.0);
	
	printf("[**] Starting kernel program 'kernel_1t1c' execution\n");
	avems = 0;
	for(int i = 0; i<2; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1c<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.2f\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 

		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	    printf("Printing first five output elements only\n");
		for(int i=0; i< 5; i++){
		printf("%.2f	", h_matC[i]);}
		printf("\n");
	    printf("[**] Average kernel execution time: %.2f\n", avems/2.0);
	
	hipFree(d_matA);
	hipFree(d_matB);
	hipFree(d_matC);
	free(h_matA);
	free(h_matB);
	free(h_matC);

	return 0;
}

